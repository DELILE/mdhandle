#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifndef LJC_GPU_KERNEL
#define LJC_GPU_KERNEL

#ifdef NV_KERNEL

#include "nv_kernel_def.h"
texture<float4> pos_tex;
texture<float> q_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int& i, const double4 *pos)
{
  return pos[i];
}
__inline double fetch_q(const int& i, const double *q)
{
  return q[i];
}
#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(pos_tex, i);
}
__inline float fetch_q(const int& i, const float *q)
{
  return tex1Dfetch(q_tex, i);
}
#endif

#else

#pragma OPENCL EXTENSION cl_khr_fp64: enable
#define GLOBAL_ID_X get_global_id(0)
#define THREAD_ID_X get_local_id(0)
#define BLOCK_ID_X get_group_id(0)
#define BLOCK_SIZE_X get_local_size(0)
#define __syncthreads() barrier(CLK_LOCAL_MEM_FENCE)
#define __inline inline

#define fetch_pos(i,y) x_[i]
#define fetch_q(i,y) q_[i]
#define BLOCK_PAIR 64
#define MAX_SHARED_TYPES 8

#endif

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp2 double2
#define numtyp4 double4
#define acctyp double
#define acctyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp double
#define acctyp4 double4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp2 float2
#define numtyp4 float4
#define acctyp float
#define acctyp4 float4
#endif

#define SBBITS 30
#define NEIGHMASK 0x3FFFFFFF
__inline int sbmask(int j) { return j >> SBBITS & 3; }

__kernel void kernel_pair(__global numtyp4 *x_, __global numtyp4 *lj1,
                          __global numtyp4* lj3, const int lj_types, 
                          __global numtyp *sp_lj_in, __global int *dev_nbor, 
                          __global int *dev_packed, __global acctyp4 *ans,
                          __global acctyp *engv, const int eflag,
                          const int vflag, const int inum,
                          const int nbor_pitch, __global numtyp *q_ ,
                          __global numtyp *cutsq, const numtyp qqrd2e,
                          const int t_per_atom) {
  int tid=THREAD_ID_X;
  int ii=mul24((int)BLOCK_ID_X,(int)(BLOCK_SIZE_X)/t_per_atom);
  ii+=tid/t_per_atom;
  int offset=tid%t_per_atom;

  __local numtyp sp_lj[8];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];
  sp_lj[4]=sp_lj_in[4];
  sp_lj[5]=sp_lj_in[5];
  sp_lj[6]=sp_lj_in[6];
  sp_lj[7]=sp_lj_in[7];

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;
  
  if (ii<inum) {
    __global int *nbor=dev_nbor+ii;
    int i=*nbor;
    nbor+=nbor_pitch;
    int numj=*nbor;
    nbor+=nbor_pitch;

    int n_stride;
    __global int *list_end;
    if (dev_nbor==dev_packed) {
      list_end=nbor+mul24(numj,nbor_pitch);
      nbor+=mul24(offset,nbor_pitch);
      n_stride=mul24(t_per_atom,nbor_pitch);
    } else {
      nbor=dev_packed+*nbor;
      list_end=nbor+numj;
      n_stride=t_per_atom;
      nbor+=offset;
    }
  
    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    numtyp qtmp=fetch_q(i,q_);
    int itype=ix.w;

    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq[mtype]) {
        numtyp r2inv=(numtyp)1.0/rsq;
        numtyp forcecoul, force_lj, force, r6inv;

        if (rsq < lj1[mtype].z) {
          r6inv = r2inv*r2inv*r2inv;
          force_lj = factor_lj*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        } else
          force_lj = (numtyp)0.0;

        if (rsq < lj1[mtype].w) 
          forcecoul = qqrd2e*qtmp*fetch_q(j,q_)*sqrt(r2inv)*factor_coul;
        else
          forcecoul = (numtyp)0.0;

        force = (force_lj + forcecoul) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          e_coul += forcecoul;
          if (rsq < lj1[mtype].z) {
            numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
            energy+=factor_lj*(e-lj3[mtype].z);
          } 
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  
  // Reduce answers
  if (t_per_atom>1) {
    __local acctyp red_acc[6][BLOCK_PAIR];
    
    red_acc[0][tid]=f.x;
    red_acc[1][tid]=f.y;
    red_acc[2][tid]=f.z;
    red_acc[3][tid]=energy;
    red_acc[4][tid]=e_coul;

    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
      if (offset < s) {
        for (int r=0; r<5; r++)
          red_acc[r][tid] += red_acc[r][tid+s];
      }
    }
    
    f.x=red_acc[0][tid];
    f.y=red_acc[1][tid];
    f.z=red_acc[2][tid];
    energy=red_acc[3][tid];
    e_coul=red_acc[4][tid];

    if (vflag>0) {
      for (int r=0; r<6; r++)
        red_acc[r][tid]=virial[r];

      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
        if (offset < s) {
          for (int r=0; r<6; r++)
            red_acc[r][tid] += red_acc[r][tid+s];
        }
      }
    
      for (int r=0; r<6; r++)
        virial[r]=red_acc[r][tid];
    }
  }

  // Store answers
  if (ii<inum && offset==0) {
    __global acctyp *ap1=engv+ii;
    if (eflag>0) {
      *ap1=energy;
      ap1+=inum;
      *ap1=e_coul;
      ap1+=inum;
    }
    if (vflag>0) {
      for (int i=0; i<6; i++) {
        *ap1=virial[i];
        ap1+=inum;
      }
    }
    ans[ii]=f;
  } // if ii
}

__kernel void kernel_pair_fast(__global numtyp4 *x_, __global numtyp4 *lj1_in,
                               __global numtyp4* lj3_in, 
                               __global numtyp* sp_lj_in,
                               __global int *dev_nbor, __global int *dev_packed,
                               __global acctyp4 *ans, __global acctyp *engv, 
                               const int eflag, const int vflag, const int inum, 
                               const int nbor_pitch, __global numtyp *q_,
                               __global numtyp *_cutsq, const numtyp qqrd2e,
                               const int t_per_atom) {
  int tid=THREAD_ID_X;
  int ii=mul24((int)BLOCK_ID_X,(int)(BLOCK_SIZE_X)/t_per_atom);
  ii+=tid/t_per_atom;
  int offset=tid%t_per_atom;

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp cutsq[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[8];
  if (tid<8)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    cutsq[tid]=_cutsq[tid];
    if (eflag>0)
      lj3[tid]=lj3_in[tid];
  }
  
  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;
  
  __syncthreads();
  
  if (ii<inum) {
    __global int *nbor=dev_nbor+ii;
    int i=*nbor;
    nbor+=nbor_pitch;
    int numj=*nbor;
    nbor+=nbor_pitch;

    int n_stride;
    __global int *list_end;
    if (dev_nbor==dev_packed) {
      list_end=nbor+mul24(numj,nbor_pitch);
      nbor+=mul24(offset,nbor_pitch);
      n_stride=mul24(t_per_atom,nbor_pitch);
    } else {
      nbor=dev_packed+*nbor;
      list_end=nbor+numj;
      n_stride=t_per_atom;
      nbor+=offset;
    }
  
    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    numtyp qtmp=fetch_q(i,q_);
    int iw=ix.w;
    int itype=mul24((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq[mtype]) {
        numtyp r2inv=(numtyp)1.0/rsq;
        numtyp forcecoul, force_lj, force, r6inv;

        if (rsq < lj1[mtype].z) {
          r6inv = r2inv*r2inv*r2inv;
          force_lj = factor_lj*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        } else
          force_lj = (numtyp)0.0;

        if (rsq < lj1[mtype].w)
          forcecoul = qqrd2e*qtmp*fetch_q(j,q_)*sqrt(r2inv)*factor_coul;
        else
          forcecoul = (numtyp)0.0;

        force = (force_lj + forcecoul) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          e_coul += forcecoul;
          if (rsq < lj1[mtype].z) {
            numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
            energy+=factor_lj*(e-lj3[mtype].z);
          }
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii

  // Reduce answers
  if (t_per_atom>1) {
    __local acctyp red_acc[6][BLOCK_PAIR];
    
    red_acc[0][tid]=f.x;
    red_acc[1][tid]=f.y;
    red_acc[2][tid]=f.z;
    red_acc[3][tid]=energy;
    red_acc[4][tid]=e_coul;

    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
      if (offset < s) {
        for (int r=0; r<5; r++)
          red_acc[r][tid] += red_acc[r][tid+s];
      }
    }
    
    f.x=red_acc[0][tid];
    f.y=red_acc[1][tid];
    f.z=red_acc[2][tid];
    energy=red_acc[3][tid];
    e_coul=red_acc[4][tid];

    if (vflag>0) {
      for (int r=0; r<6; r++)
        red_acc[r][tid]=virial[r];

      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {
        if (offset < s) {
          for (int r=0; r<6; r++)
            red_acc[r][tid] += red_acc[r][tid+s];
        }
      }
    
      for (int r=0; r<6; r++)
        virial[r]=red_acc[r][tid];
    }
  }

  // Store answers
  if (ii<inum && offset==0) {
    __global acctyp *ap1=engv+ii;
    if (eflag>0) {
      *ap1=energy;
      ap1+=inum;
      *ap1=e_coul;
      ap1+=inum;
    }
    if (vflag>0) {
      for (int i=0; i<6; i++) {
        *ap1=virial[i];
        ap1+=inum;
      }
    }
    ans[ii]=f;
  } // if ii*/
}

#endif

