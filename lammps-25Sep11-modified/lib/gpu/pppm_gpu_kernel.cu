#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS-Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifndef PPPM_GPU_KERNEL
#define PPPM_GPU_KERNEL

#ifdef NV_KERNEL

#include "geryon/ucl_nv_kernel.h"
texture<float4> pos_tex;
texture<float> q_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int& i, const double4 *pos)
{
  return pos[i];
}
__inline double fetch_q(const int& i, const double *q)
{
  return q[i];
}

#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(pos_tex, i);
}
__inline float fetch_q(const int& i, const float *q)
{
  return tex1Dfetch(q_tex, i);
}

#endif

// Allow PPPM to compile without atomics for NVIDIA 1.0 cards, error
// generated at runtime with use of pppm/gpu
#if (__CUDA_ARCH__ < 110)
#define atomicAdd(x,y) *(x)+=0
#endif

#else

#pragma OPENCL EXTENSION cl_khr_fp64: enable
#pragma OPENCL EXTENSION cl_khr_global_int32_base_atomics: enable
#define GLOBAL_ID_X get_global_id(0)
#define THREAD_ID_X get_local_id(0)
#define BLOCK_ID_X get_group_id(0)
#define BLOCK_SIZE_X get_local_size(0)
#define GLOBAL_SIZE_X get_global_size(0)
#define __syncthreads() barrier(CLK_LOCAL_MEM_FENCE)
#define __inline inline

#define fetch_pos(i,y) x_[i]
#define fetch_q(i,y) q_[i]
#define MEM_THREADS 16

#endif

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp4 double4
#define acctyp double
#define acctyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp4 float4
#define acctyp double
#define acctyp4 double4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp4 float4
#define acctyp float
#define acctyp4 float4
#endif

// Maximum order for spline
#define PPPM_MAX_SPLINE 8
// Thread block size for PPPM kernels
// - Must be >=PPPM_MAX_SPLINE^2
// - Must be a multiple of 32
#define PPPM_BLOCK_1D 64
// Number of threads per pencil for charge spread
#define PENCIL_SIZE MEM_THREADS
// Number of pencils per block for charge spread
#define BLOCK_PENCILS (PPPM_BLOCK_1D/PENCIL_SIZE)

__kernel void particle_map(__global numtyp4 *x_,  __global numtyp *q_,
                           const grdtyp delvolinv, const int nlocal, 
                           __global int *counts, __global grdtyp4 *ans, 
                           const grdtyp b_lo_x, const grdtyp b_lo_y,
                           const grdtyp b_lo_z, const grdtyp delxinv,
                           const grdtyp delyinv, const grdtyp delzinv,
                           const int nlocal_x, const int nlocal_y,
                           const int nlocal_z, const int atom_stride,
                           const int max_atoms, __global int *error) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;

  // Resequence the atom indices to avoid collisions during atomic ops
  int nthreads=GLOBAL_SIZE_X;
  ii=mul24(ii,PPPM_BLOCK_1D);
  ii-=(ii/nthreads)*(nthreads-1);

  int nx,ny,nz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);
    grdtyp4 delta;
    delta.w=delvolinv*fetch_q(ii,q_);
    
    if (delta.w!=(grdtyp)0.0) {
      delta.x=(p.x-b_lo_x)*delxinv;
      nx=delta.x;
      delta.y=(p.y-b_lo_y)*delyinv;
      ny=delta.y;
      delta.z=(p.z-b_lo_z)*delzinv;
      nz=delta.z;

      if (delta.x<(grdtyp)0 || delta.y<(grdtyp)0 || delta.z<(grdtyp)0 || 
          nx>=nlocal_x || ny>=nlocal_y || nz>=nlocal_z)
        *error=1;
      else {
        delta.x=nx+(grdtyp)0.5-delta.x;
        delta.y=ny+(grdtyp)0.5-delta.y;
        delta.z=nz+(grdtyp)0.5-delta.z;
      
        int i=nz*nlocal_y*nlocal_x+ny*nlocal_x+nx;
        int old=atom_add(counts+i, 1);
        if (old>=max_atoms) {
          *error=2;
          atom_add(counts+i, -1);
        } else
          ans[atom_stride*old+i]=delta;
      }
    }
  }
}

/* --------------------------- */

__kernel void make_rho(__global int *counts, __global grdtyp4 *atoms,
                       __global grdtyp *brick, __global grdtyp *_rho_coeff,
                       const int atom_stride, const int npts_x,
                       const int npts_y, const int npts_z, const int nlocal_x,
                       const int nlocal_y, const int nlocal_z,
                       const int order_m_1, const int order, const int order2) {
  __local grdtyp rho_coeff[PPPM_MAX_SPLINE*PPPM_MAX_SPLINE];
  __local grdtyp front[BLOCK_PENCILS][PENCIL_SIZE+PPPM_MAX_SPLINE];
  __local grdtyp ans[PPPM_MAX_SPLINE][PPPM_BLOCK_1D];
  
  int tid=THREAD_ID_X;
  if (tid<order2+order)
    rho_coeff[tid]=_rho_coeff[tid];
    
  int pid=tid/PENCIL_SIZE;
  int fid=tid%PENCIL_SIZE;
  int fid_halo=PENCIL_SIZE+fid;
  if (fid<order) 
    front[pid][fid_halo]=(grdtyp)0.0;

  __syncthreads();

  int bt=BLOCK_ID_X*BLOCK_PENCILS+pid;
  int ny=bt%npts_y;
  int nz=bt/npts_y;
  int y_start=0;
  int z_start=0;
  int y_stop=order;
  int z_stop=order;
  if (ny<order_m_1)
    y_start=order_m_1-ny;
  if (nz<order_m_1)
    z_start=order_m_1-nz;
  if (ny>=nlocal_y)
    y_stop-=ny-nlocal_y+1;
  if (nz>=nlocal_z)
    z_stop-=nz-nlocal_z+1;
  int z_stride=mul24(nlocal_x,nlocal_y);

  int loop_count=npts_x/PENCIL_SIZE+1;
  int nx=fid;
  int pt=mul24(nz,mul24(npts_y,npts_x))+mul24(ny,npts_x)+nx;
  for (int i=0 ; i<loop_count; i++) {
    for (int n=0; n<order; n++)
      ans[n][tid]=(grdtyp)0.0;
    if (nx<nlocal_x && nz<npts_z) {
      int z_pos=mul24(nz+z_start-order_m_1,z_stride);
      for (int m=z_start; m<z_stop; m++) {
        int y_pos=mul24(ny+y_start-order_m_1,nlocal_x);
        for (int l=y_start; l<y_stop; l++) {
          int pos=z_pos+y_pos+nx;
          int natoms=mul24(counts[pos],atom_stride);
          for (int row=pos; row<natoms; row+=atom_stride) {
            grdtyp4 delta=atoms[row];
      
            grdtyp rho1d_1=(grdtyp)0.0;
            grdtyp rho1d_2=(grdtyp)0.0;
            for (int k=order2+order-1; k > -1; k-=order) {
              rho1d_1=rho_coeff[k-l]+rho1d_1*delta.y;
              rho1d_2=rho_coeff[k-m]+rho1d_2*delta.z;
            }
            delta.w*=rho1d_1*rho1d_2;

            for (int n=0; n<order; n++) {
              grdtyp rho1d_0=(grdtyp)0.0;
              for (int k=order2+n; k>=n; k-=order)
                rho1d_0=rho_coeff[k]+rho1d_0*delta.x;
              ans[n][tid]+=delta.w*rho1d_0;
            }
          }
          y_pos+=nlocal_x;
        }
        z_pos+=z_stride;
      }
    }
    
    __syncthreads();
    if (fid<order) {
      front[pid][fid]=front[pid][fid_halo];
      front[pid][fid_halo]=(grdtyp)0.0;
    } else 
      front[pid][fid]=(grdtyp)0.0;
    
    for (int n=0; n<order; n++) {
      front[pid][fid+n]+=ans[n][tid];
      __syncthreads();
    }

    if (nx<npts_x && nz<npts_z)
      brick[pt]=front[pid][fid];
    pt+=PENCIL_SIZE;
    nx+=PENCIL_SIZE;
  }
}

__kernel void interp(__global numtyp4 *x_, __global numtyp *q_,
                     const int nlocal, __global grdtyp4 *brick,
                     __global grdtyp *_rho_coeff, const int npts_x,
                     const int npts_yx, const grdtyp b_lo_x,
                     const grdtyp b_lo_y, const grdtyp b_lo_z,
                     const grdtyp delxinv,  const grdtyp delyinv,
                     const grdtyp delzinv, const int order,
                     const int order2, const grdtyp qqrd2e_scale, 
                     __global acctyp4 *ans) {
  __local grdtyp rho_coeff[PPPM_MAX_SPLINE*PPPM_MAX_SPLINE];
  __local grdtyp rho1d_0[PPPM_MAX_SPLINE][PPPM_BLOCK_1D];
  __local grdtyp rho1d_1[PPPM_MAX_SPLINE][PPPM_BLOCK_1D];

  int tid=THREAD_ID_X;
  if (tid<order2+order)
    rho_coeff[tid]=_rho_coeff[tid];
  __syncthreads();
  
  int ii=tid+BLOCK_ID_X*BLOCK_SIZE_X;
  
  int nx,ny,nz;
  grdtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p=fetch_pos(ii,x_);
    grdtyp qs=qqrd2e_scale*fetch_q(ii,q_);

    acctyp4 ek;
    ek.x=(acctyp)0.0;
    ek.y=(acctyp)0.0;
    ek.z=(acctyp)0.0;
    if (qs!=(grdtyp)0.0) {
      tx=(p.x-b_lo_x)*delxinv;
      nx=tx;
      ty=(p.y-b_lo_y)*delyinv;
      ny=ty;
      tz=(p.z-b_lo_z)*delzinv;
      nz=tz;

      grdtyp dx=nx+(grdtyp)0.5-tx;
      grdtyp dy=ny+(grdtyp)0.5-ty;
      grdtyp dz=nz+(grdtyp)0.5-tz;

      for (int k=0; k<order; k++) {
        rho1d_0[k][tid]=(grdtyp)0.0;
        rho1d_1[k][tid]=(grdtyp)0.0;
        for (int l=order2+k; l>=k; l-=order) {
          rho1d_0[k][tid]=rho_coeff[l]+rho1d_0[k][tid]*dx;
          rho1d_1[k][tid]=rho_coeff[l]+rho1d_1[k][tid]*dy;
        }
      }
        
      int mz=mul24(nz,npts_yx)+nx;
      for (int n=0; n<order; n++) {
        grdtyp rho1d_2=(grdtyp)0.0;
        for (int k=order2+n; k>=n; k-=order)
          rho1d_2=rho_coeff[k]+rho1d_2*dz;
        grdtyp z0=qs*rho1d_2;
        int my=mz+mul24(ny,npts_x);
        for (int m=0; m<order; m++) {
          grdtyp y0=z0*rho1d_1[m][tid];
  	      for (int l=0; l<order; l++) {
  	        grdtyp x0=y0*rho1d_0[l][tid];
  	        grdtyp4 el=brick[my+l];
  	        ek.x-=x0*el.x;
  	        ek.y-=x0*el.y;
  	        ek.z-=x0*el.z;
  	      }
          my+=npts_x;
        }
        mz+=npts_yx;
  	  }
    }
    ans[ii]=ek;
	}
}

#endif

