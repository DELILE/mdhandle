#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Peng Wang (Nvidia), penwang@nvidia.com
                         Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifdef NV_KERNEL

#include "nv_kernel_def.h"
texture<float4> neigh_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int i, const double4 *pos)
{
  return pos[i];
}
#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(neigh_tex, i);
}
#endif

#else

#define fetch_pos(i,y) x_[i]
#define BLOCK_NBOR_BUILD 64

#endif

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp4 float4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp4 float4
#endif

#define BLOCK_CELL_2D 8

#define SBBITS 30

#define SBBITS 30

__kernel void transpose(int *out, int *in, int columns_in, int rows_in)
{
	__local float block[BLOCK_CELL_2D][BLOCK_CELL_2D+1];
	
	unsigned ti=THREAD_ID_X;
	unsigned tj=THREAD_ID_Y;
	unsigned bi=BLOCK_ID_X;
	unsigned bj=BLOCK_ID_Y;
	
	unsigned i=bi*BLOCK_CELL_2D+ti;
	unsigned j=bj*BLOCK_CELL_2D+tj;
	if ((i<columns_in) && (j<rows_in))
		block[tj][ti]=in[j*columns_in+i];

	__syncthreads();

	i=bj*BLOCK_CELL_2D+ti;
	j=bi*BLOCK_CELL_2D+tj;
	if ((i<rows_in) && (j<columns_in))
		out[j*rows_in+i] = block[ti][tj];
}

__kernel void calc_cell_id(numtyp4 *pos, unsigned *cell_id, int *particle_id,
                           numtyp boxlo0, 
                           numtyp boxlo1, numtyp boxlo2, numtyp boxhi0, 
                           numtyp boxhi1, numtyp boxhi2, numtyp cell_size, 
                           int ncellx, int ncelly, int nall) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i < nall) {
    numtyp4 p = fetch_pos(i,pos); //pos[i];

    p.x -= boxlo0;
    p.y -= boxlo1;
    p.z -= boxlo2;
    
    p.x = fmaxf(p.x, -cell_size);
    p.x = fminf(p.x, boxhi0-boxlo0+cell_size);
    p.y = fmaxf(p.y, -cell_size);
    p.y = fminf(p.y, boxhi1-boxlo1+cell_size);
    p.z = fmaxf(p.z, -cell_size);
    p.z = fminf(p.z, boxhi2-boxlo2+cell_size);
    
    unsigned int id = (unsigned int)(p.x/cell_size + 1.0) 
      + (unsigned int)(p.y/cell_size + 1.0) * ncellx
      + (unsigned int)(p.z/cell_size + 1.0) * ncellx * ncelly;
    
    cell_id[i] = id;
    particle_id[i] = i;
  }
}

__kernel void kernel_calc_cell_counts(unsigned *cell_id,
                                      int *cell_counts, int nall, int ncell) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nall) {
    int id = cell_id[idx];

    // handle boundary cases
    if (idx == 0) {
      for (int i = 0; i < id + 1; i++) 
        cell_counts[i] = 0;
    }
    if (idx == nall - 1) {
      for (int i = id+1; i <= ncell; i++) 
        cell_counts[i] = nall;
    }

    if (idx > 0 && idx < nall) {
      int id_l = cell_id[idx-1];
      if (id != id_l) {
        for (int i = id_l+1; i <= id; i++) 
          cell_counts[i] = idx;
      }
    }
  }
}

__kernel void calc_neigh_list_cell(numtyp4 *pos,
				     int *cell_particle_id, 
				     int *cell_counts,
				     int *nbor_list,
				     int *host_nbor_list,
				     int *host_numj, 
				     int neigh_bin_size, 
				     numtyp cell_size,
				     int ncellx, int ncelly, int ncellz,
				     int inum, int nt, int nall)
{
  int tid = threadIdx.x;
  int ix = blockIdx.x;
  int iy = blockIdx.y % ncelly;
  int iz = blockIdx.y / ncelly;
	  
  int icell = ix + iy*ncellx + iz*ncellx*ncelly;

  __shared__ int cell_list_sh[BLOCK_NBOR_BUILD];
  __shared__ numtyp4 pos_sh[BLOCK_NBOR_BUILD];

  int icell_begin = cell_counts[icell];
  int icell_end = cell_counts[icell+1];

  int nborz0 = max(iz-1,0), nborz1 = min(iz+1, ncellz-1),
      nbory0 = max(iy-1,0), nbory1 = min(iy+1, ncelly-1),
      nborx0 = max(ix-1,0), nborx1 = min(ix+1, ncellx-1);

  numtyp4 diff;
  numtyp r2;
  for (int ii = 0; ii < ceil((numtyp)(icell_end - icell_begin)/blockDim.x); ii++) {
    int i = icell_begin + tid + ii*blockDim.x;
    int pid_i = nall, pid_j, stride;
    numtyp4 atom_i, atom_j;
    int cnt = 0;    
    int *neigh_counts, *neigh_list;
    
    if (i < icell_end)
      pid_i = cell_particle_id[i];

    if (pid_i < nt) {
      atom_i = fetch_pos(pid_i,pos); //pos[pid_i];
    }
    if (pid_i < inum) {
      stride=inum;
      neigh_counts=nbor_list+stride+pid_i;
      neigh_list=neigh_counts+stride;
      nbor_list[pid_i]=pid_i;
    } else {
      stride=1;
    	neigh_counts=host_numj+pid_i-inum;
      neigh_list=host_nbor_list+(pid_i-inum)*neigh_bin_size;
    }
    
    // loop through neighbors

    for (int nborz = nborz0; nborz <= nborz1; nborz++) {
      for (int nbory = nbory0; nbory <= nbory1; nbory++) {
        for (int nborx = nborx0; nborx <= nborx1; nborx++) {
	
          int jcell = nborx + nbory*ncellx + nborz*ncellx*ncelly;
		
          int jcell_begin = cell_counts[jcell];
          int jcell_end = cell_counts[jcell+1];
          int num_atom_cell = jcell_end - jcell_begin;
	  
          // load jcell to shared memory
          int num_iter = (int)ceil((numtyp)num_atom_cell/BLOCK_NBOR_BUILD);

          for (int k = 0; k < num_iter; k++) {
            int end_idx = min(BLOCK_NBOR_BUILD, num_atom_cell-k*BLOCK_NBOR_BUILD);
	    
            if (tid < end_idx) {
              pid_j =  cell_particle_id[tid+k*BLOCK_NBOR_BUILD+jcell_begin];
              cell_list_sh[tid] = pid_j;
              atom_j = fetch_pos(pid_j,pos); //[pid_j];
              pos_sh[tid].x = atom_j.x;
              pos_sh[tid].y = atom_j.y;
              pos_sh[tid].z = atom_j.z;
            }
            __syncthreads();
	    
            if (pid_i < nt) {
	    
              for (int j = 0; j < end_idx; j++) {
                int pid_j = cell_list_sh[j]; // gather from shared memory
                diff.x = atom_i.x - pos_sh[j].x;
                diff.y = atom_i.y - pos_sh[j].y;
                diff.z = atom_i.z - pos_sh[j].z;
		
                r2 = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
                if (r2 < cell_size*cell_size && r2 > 1e-5) {
                  if (cnt < neigh_bin_size) {
                    *neigh_list = pid_j;
                    neigh_list+=stride;
                  }
                  cnt++;
                }		
              }
            }
	          __syncthreads();
	        } // for (k)
        }
      }
    }
    if (pid_i < nt)
      *neigh_counts = cnt;
  } // for (i)
}

__kernel void kernel_special(__global int *dev_nbor, 
                             __global int *host_nbor_list, 
                             __global int *host_numj, __global int *tag,
                             __global int *nspecial, __global int *special,
                             int inum, int nt, int max_nbors) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;

  if (ii<nt) {
    int stride;
    __global int *list, *list_end;
    
    int n1=nspecial[ii*3];
    int n2=nspecial[ii*3+1];
    int n3=nspecial[ii*3+2];

    int numj;
    if (ii < inum) {
      stride=inum;
      list=dev_nbor+stride+ii;
      numj=*list;
      list+=stride;
    } else {
      stride=1;
      list=host_nbor_list+(ii-inum)*max_nbors;
      numj=host_numj[ii-inum];
    }
    list_end=list+numj*stride;
  
    for ( ; list<list_end; list+=stride) {
      int nbor=*list;
      int jtag=tag[nbor];

      int offset=ii;
      for (int i=0; i<n3; i++) {
        if (special[offset]==jtag) {
          int which = 1;
          if (i>=n1)
            which++;
          if (i>=n2)
            which++;
          nbor=nbor ^ (which << SBBITS);
          *list=nbor;
        }
        offset+=nt;
      }
    }
  } // if ii
}
