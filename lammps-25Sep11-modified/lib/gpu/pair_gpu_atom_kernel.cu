#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifdef NV_KERNEL
#include "geryon/ucl_nv_kernel.h"
#else
#pragma OPENCL EXTENSION cl_khr_fp64: enable
#define GLOBAL_ID_X get_global_id(0)
#endif

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp4 double4
#else
#define numtyp float
#define numtyp4 float4
#endif

__kernel void kernel_cast_x(__global numtyp4 *x_type, __global double *x,
                            __global int *type, const int nall) {
  int ii=GLOBAL_ID_X;

  if (ii<nall) {
    numtyp4 xt;
    xt.w=type[ii];
    int i=ii*3;
    xt.x=x[i];
    xt.y=x[i+1];
    xt.z=x[i+2];
    x_type[ii]=xt;
  } // if ii
}
